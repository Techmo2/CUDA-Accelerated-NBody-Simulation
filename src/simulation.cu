#pragma once
#include <cstdint>
#include <hip/hip_runtime.h>

#include <ctime>
#include <iostream>
#include <cstdlib>
#include <chrono>
#include <iomanip>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

typedef struct {
	float x;
	float y;
	float z;
} vec3;

class Body {
public:
	bool blank;
	vec3 position;
	vec3 velocity;
	float mass;
	float radius;
	uint8_t* color;
	Body();
	Body(vec3 position, float mass, float radius);
	Body(vec3 position, vec3 velocity, float mass, float radius);
	void setColor(uint8_t r, uint8_t g, uint8_t b);
};

Body::Body() {
	this->blank = true;
}

Body::Body(vec3 position, float mass, float radius) {
	vec3 vel;
	vel.x = 0;
	vel.y = 0;
	vel.z = 0;

	this->position = position;
	this->velocity = vel;
	this->mass = mass;
	this->radius = radius;
	this->color = new uint8_t[3]{ 200, 0, 200 };
	this->blank = false;
}

Body::Body(vec3 position, vec3 velocity, float mass, float radius) {
	this->position = position;
	this->velocity = velocity;
	this->mass = mass;
	this->radius = radius;
	this->color = new uint8_t[3]{ 255, 0, 255 };
	this->blank = false;
}

void Body::setColor(uint8_t r, uint8_t g, uint8_t b) {
	this->color = new uint8_t[3]{ r, g, b };
}

__global__
void step(Body* bodiesIn, Body* results, int n, float dt) {
	const int i = threadIdx.x;

	if (i < n) {
		Body a = bodiesIn[i];
		float EPS = 3e4;
		float G = 1.0;
		float fx = 0;
		float fy = 0;
		float fz = 0;

		for (int j = 0; j < n; j++) {
			if (j != i) {
				Body b = bodiesIn[j];

				float dx = b.position.x - a.position.x;
				float dy = b.position.y - a.position.y;
				float dz = b.position.z - a.position.z;
				float dist = sqrt(dx*dx + dy*dy + dz*dz);

				float F = (G * a.mass * b.mass) / (dist * dist + EPS * EPS);
				fx += F * dx / dist;
				fy += F * dy / dist;
				fz += F * dz / dist;
			}
		}

		a.velocity.x += dt * fx / a.mass;
		a.velocity.y += dt * fy / a.mass;
		a.velocity.z += dt * fz / a.mass;
		a.position.x += dt * a.velocity.x;
		a.position.y += dt * a.velocity.y;
		a.position.z += dt * a.velocity.z;

		results[i] = a;
	}
}

__global__
void prepareForNextStep(Body* bodiesIn, Body* results, int n) {
	const int i = threadIdx.x;

	if (i < n) {
		bodiesIn[i] = results[i];
	}
}


class Simulation {
public:
	Simulation(unsigned int maxBodies, const int numThreads);
	void addBody(Body b);
	void addBody(vec3 position, vec3 velocity, float mass, float radius);
	void sendBodiesToDevice();
	void readBodiesFromDevice();
	void stepSimulation(float dt);
	Body getBody(int index);
	void cleanup();

private:
	unsigned int currentStep;
	unsigned int maxBodies;
	unsigned int numBodies;
	int numThreads;
	Body* bodies;
	Body* inBodies;
	Body* resBodies;
};

Simulation::Simulation(unsigned int maxBodies, const int numThreads) {
	this->maxBodies = maxBodies;
	this->bodies = new Body[maxBodies];
	this->numBodies = 0;
	this->currentStep = 0;
	this->numThreads = numThreads;
}

void Simulation::addBody(Body b) {
	if (numBodies < maxBodies) {
		bodies[numBodies] = b;
		numBodies++;
	}
	
}

void Simulation::addBody(vec3 position, vec3 velocity, float mass, float radius) {
	if (numBodies < maxBodies) {
		bodies[numBodies] = Body(position, velocity, mass, radius);
		numBodies++;
	}
}

void Simulation::stepSimulation(float dt) {
	if (currentStep == 0) {
		step<<< 1, numThreads >>>(inBodies, resBodies, numBodies, 1.0);
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
		currentStep++;
	}
	else {
		prepareForNextStep<<< 1, numThreads >>>(inBodies, resBodies, numBodies);
		step<<< 1, numThreads >>>(inBodies, resBodies, numBodies, 1.0);
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
		currentStep++;
	}
}

// Only need to do this once
// Copy bodies from bodies array to gpu, and allocate space for result bodies on gpu
void Simulation::sendBodiesToDevice() {
	hipMalloc(&inBodies, sizeof(Body) * maxBodies);
	hipMalloc(&resBodies, sizeof(Body) * maxBodies);
	hipMemcpy(inBodies, bodies, sizeof(Body) * maxBodies, hipMemcpyHostToDevice);
}

// Copy current simulation step from gpu to bodies array
void Simulation::readBodiesFromDevice() {
	hipMemcpy(bodies, resBodies, sizeof(Body) * maxBodies, hipMemcpyDeviceToHost);
}

Body Simulation::getBody(int index) {
	return bodies[index];
}

void Simulation::cleanup() {
	hipFree(inBodies);

	//delete[] bodies;
	//delete[] resBodies;

}


int main(int argc, char** argv) {
	int maxBodies = 1000;
	int threads = 256;
	int cycles = 1000;

	if(argc == 4){
	std::cout << "Starting simulation with " << atoi(argv[1]) << " bodies and " << atoi(argv[2]) << " threads" << std::endl;
	int maxBodies = atoi(argv[1]);
	int threads = atoi(argv[2]);
	int cycles = atoi(argv[3]);
	}
	else{
		std::cout << "No parameters given, starting with 1000 bodies running on 256 threads" << std::endl;
	}

	Simulation* sim = new Simulation(maxBodies, threads);
	srand(time(NULL));

	for (int i = 0; i < maxBodies; i++) {
		vec3 pos;
		vec3 vel;

		pos.x = rand() % 10000;
		pos.y = rand() % 10000;
		pos.z = rand() % 10000;

		vel.x = 0;
		vel.y = 0;
		vel.z = 0;
		
		sim->addBody(pos, vel, 10000000 - rand() % 1000, 100);
	}

	std::cout << "Bodies added, moving to gpu memory" << std::endl;
	std::cout << "6th body: ";

	std::cout << std::fixed;
	std::cout << std::setprecision(2);

	Body c = sim->getBody(5);
	std::cout << " x-" << c.position.x << " y-" << c.position.y << " z-" << c.position.z << std::endl;

	sim->sendBodiesToDevice();

	std::cout << "Gpu memory populated, stepping simulation " << cycles << " times" << std::endl;

	auto start = std::chrono::high_resolution_clock::now();
	for (int i = 0; i < cycles; i++) {
		sim->stepSimulation(1.0);
	}
	auto finish = std::chrono::high_resolution_clock::now();
	long long elapsed = std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count();
	float elapsed_seconds = elapsed / 1000000000.0;
	float rate = (float)(cycles * maxBodies) / elapsed_seconds;

	std::cout << "1000 steps done, " << elapsed_seconds << " seconds elapsed, reading results" << std::endl;

	sim->readBodiesFromDevice();

	std::cout << "6th result: ";

	Body b = sim->getBody(5);
	vec3 bpos = b.position;

	std::cout << "x-" << bpos.x << " y-" << bpos.y << " z-" << bpos.z << std::endl;
	std::cout << cycles * maxBodies << " bodies processed in " << elapsed_seconds << " seconds (" << rate << " bodies per second)" << std::endl;

	sim->cleanup();
	return 0;
}