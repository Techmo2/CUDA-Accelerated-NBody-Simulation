#include <cstdint>
#include <hip/hip_runtime.h>

#include <ctime>
#include <iostream>
#include <cstdlib>
#include <chrono>
#include <iomanip>
#include <fstream>
#include <string>
#include <sstream>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

typedef struct {
	float x;
	float y;
	float z;
} vec3;

class Body {
public:
	bool blank;
	vec3 position;
	vec3 velocity;
	float mass;
	float radius;
	uint8_t* color;
	Body();
	Body(vec3 position, float mass, float radius);
	Body(vec3 position, vec3 velocity, float mass, float radius);
	void setColor(uint8_t r, uint8_t g, uint8_t b);
};

Body::Body() {
	this->blank = true;
}

Body::Body(vec3 position, float mass, float radius) {
	vec3 vel;
	vel.x = 0;
	vel.y = 0;
	vel.z = 0;

	this->position = position;
	this->velocity = vel;
	this->mass = mass;
	this->radius = radius;
	this->color = new uint8_t[3]{ 200, 0, 200 };
	this->blank = false;
}

Body::Body(vec3 position, vec3 velocity, float mass, float radius) {
	this->position = position;
	this->velocity = velocity;
	this->mass = mass;
	this->radius = radius;
	this->color = new uint8_t[3]{ 255, 0, 255 };
	this->blank = false;
}

void Body::setColor(uint8_t r, uint8_t g, uint8_t b) {
	this->color = new uint8_t[3]{ r, g, b };
}

__global__
void step(Body* bodiesIn, Body* results, int n, float dt) {
	for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x){
		Body a = bodiesIn[i];
		float EPS = 3e4;
		float G = 1.0;
		float fx = 0;
		float fy = 0;
		float fz = 0;

		for (int j = 0; j < n; j++) {
			if (j != i) {
				Body b = bodiesIn[j];

				float dx = b.position.x - a.position.x;
				float dy = b.position.y - a.position.y;
				float dz = b.position.z - a.position.z;
				float dist = sqrt(dx*dx + dy*dy + dz*dz);

				float F = (G * a.mass * b.mass) / (dist * dist + EPS * EPS);
				fx += F * dx / dist;
				fy += F * dy / dist;
				fz += F * dz / dist;
			}
		}

		a.velocity.x += dt * fx / a.mass;
		a.velocity.y += dt * fy / a.mass;
		a.velocity.z += dt * fz / a.mass;
		a.position.x += dt * a.velocity.x;
		a.position.y += dt * a.velocity.y;
		a.position.z += dt * a.velocity.z;

		results[i] = a;
	}
	
}

__global__
void prepareForNextStep(Body* bodiesIn, Body* results, int n) {
	for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x){
		bodiesIn[i] = results[i];
	}
}


class Simulation {
public:
	Simulation(unsigned int maxBodies, const int numThreads);
	void addBody(Body b);
	void addBody(vec3 position, vec3 velocity, float mass, float radius);
	void sendBodiesToDevice();
	void readBodiesFromDevice();
	void stepSimulation(float dt);
	Body getBody(int index);
	void cleanup();
	void enableFrameRecord();

private:
	unsigned int currentStep;
	unsigned int maxBodies;
	unsigned int numBodies;
	int numThreads;
	int numSMs;
	bool recordFrames;
	Body* bodies;
	Body* inBodies;
	Body* resBodies;
	std::ofstream recordFile;
	void recordFrame();
};

Simulation::Simulation(unsigned int maxBodies, const int numThreads) {
	this->maxBodies = maxBodies;
	this->bodies = new Body[maxBodies];
	this->numBodies = 0;
	this->currentStep = 0;
	this->numThreads = numThreads;
	this->recordFrames = false;
	this->numSMs = 0;

	hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);
}

void Simulation::addBody(Body b) {
	if (numBodies < maxBodies) {
		bodies[numBodies] = b;
		numBodies++;
	}
	
}

void Simulation::addBody(vec3 position, vec3 velocity, float mass, float radius) {
	if (numBodies < maxBodies) {
		bodies[numBodies] = Body(position, velocity, mass, radius);
		numBodies++;
	}
}

void Simulation::stepSimulation(float dt) {
	if (currentStep == 0) {
		step<<< 32 * numSMs, numThreads >>>(inBodies, resBodies, numBodies, 1.0);
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
		currentStep++;
	}
	else {
		prepareForNextStep<<< 32*numSMs, numThreads >>>(inBodies, resBodies, numBodies);
		step<<< 32*numSMs, numThreads >>>(inBodies, resBodies, numBodies, 1.0);
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
		currentStep++;
	}
	if(recordFrames){
		recordFrame();
	}
}

// Only need to do this once
// Copy bodies from bodies array to gpu, and allocate space for result bodies on gpu
void Simulation::sendBodiesToDevice() {
	hipMalloc(&inBodies, sizeof(Body) * maxBodies);
	hipMalloc(&resBodies, sizeof(Body) * maxBodies);
	hipMemcpy(inBodies, bodies, sizeof(Body) * maxBodies, hipMemcpyHostToDevice);
}

// Copy current simulation step from gpu to bodies array
void Simulation::readBodiesFromDevice() {
	hipMemcpy(bodies, resBodies, sizeof(Body) * maxBodies, hipMemcpyDeviceToHost);
}

Body Simulation::getBody(int index) {
	return bodies[index];
}

void Simulation::cleanup() {
	hipFree(inBodies);

	//delete[] bodies;
	//delete[] resBodies;

}

void Simulation::enableFrameRecord(){
	int err = system("rm -r frames");
	if (-1 == err)
	{
    	printf("Error creating directory!n");
    	exit(1);
	}

	err = system("mkdir -p frames");
	if (-1 == err)
	{
    	printf("Error creating directory!n");
    	exit(1);
	}

	recordFrames = true;
}

void Simulation::recordFrame(){
	readBodiesFromDevice();
	std::stringstream num;
	num << "frames/" << currentStep << ".bdat";
	recordFile.open(num.str().c_str());

	if(recordFile.is_open()){
	for(int l = 0; l < maxBodies; l++){
		Body b = this->getBody(l);
		float px = b.position.x;
		float py = b.position.y;
		float pz = b.position.z;
		recordFile << l << " " << px << " " << py << " " << pz << " " << "\n";
	}
	}
	recordFile.flush();
	recordFile.close();
}

int main(int argc, char** argv) {
	int maxBodies = 1000;
	int threads = 256;
	int cycles = 1000;
	bool record = false;

	if(argc >= 4){
	std::cout << "Starting simulation with " << atoi(argv[1]) << " bodies and " << atoi(argv[2]) << " threads" << std::endl;
		maxBodies = atoi(argv[1]);
		threads = atoi(argv[2]);
		cycles = atoi(argv[3]);
	}
	else{
		std::cout << "No parameters given, starting with 1000 bodies running on 256 threads" << std::endl;
	}

	if(argc == 5 && atoi(argv[4]) == 1){
		std::cout << "Enabled data recording, frame data will be stored in the 'frames' directory" << std::endl;
		record = true;
	}

	Simulation* sim = new Simulation(maxBodies, threads);
	if(record){
	sim -> enableFrameRecord();
	}
	srand(time(NULL));

	for (int i = 0; i < maxBodies; i++) {
		vec3 pos;
		vec3 vel;

		pos.x = rand() % 10000;
		pos.y = rand() % 10000;
		pos.z = rand() % 10000;

		vel.x = 0;
		vel.y = 0;
		vel.z = 0;
		
		sim->addBody(pos, vel, 1000000 - rand() % 1000, 100);
	}

	std::cout << "Bodies added, moving to gpu memory" << std::endl;
	std::cout << "6th body: ";

	std::cout << std::fixed;
	std::cout << std::setprecision(2);

	Body c = sim->getBody(5);
	std::cout << " x-" << c.position.x << " y-" << c.position.y << " z-" << c.position.z << std::endl;

	sim->sendBodiesToDevice();

	std::cout << "Gpu memory populated, stepping simulation " << cycles << " times" << std::endl;

	auto start = std::chrono::high_resolution_clock::now();
	for (int i = 0; i < cycles; i++) {
		sim->stepSimulation(1.0);
	}
	auto finish = std::chrono::high_resolution_clock::now();
	long long elapsed = std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count();
	double elapsed_seconds = elapsed / 1000000000.0;
	long long rate = (long long)(cycles * maxBodies) / elapsed_seconds;
	long long totalBodies = cycles * maxBodies;

	std::cout << cycles << " cycles done, " << elapsed_seconds << " seconds elapsed, reading results" << std::endl;

	sim->readBodiesFromDevice();

	std::cout << "6th result: ";

	Body b = sim->getBody(5);
	vec3 bpos = b.position;

	std::cout << "x-" << bpos.x << " y-" << bpos.y << " z-" << bpos.z << std::endl;
	std::cout << totalBodies << " bodies processed (total over " << cycles << " cycles) in " << elapsed_seconds << " seconds (" << (long)rate << " bodies per second)" << std::endl;

	sim->cleanup();
	return 0;
}